// OpenACC Vector Addition
// Hard deadline : Thu 9 Apr 2015 7:00 AM CST
#include <wb.h> 

int main(int argc, char **argv) {
  wbArg_t args;
  int i;
  int inputLength;
  float *restrict hostInput1;
  float *restrict hostInput2;
  float *restrict hostOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  #pragma acc parallel loop copyin(hostInput1[0:inputLength]) copyin(hostInput2[0:inputLength]) copyout(hostOutput[0:inputLength])
  for(i=0;i<inputLength;i++)
  {
	hostOutput[i]=hostInput1[i]+hostInput2[i];
  }  

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
